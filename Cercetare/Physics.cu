#include "Physics.cuh"
#include "SimulationSettings.cuh"
#include "Kernels.cuh"
#include <iostream>
#include <chrono>

Physics::Physics() {}

Physics::~Physics() {}

void Physics::Step(GameState& gameState, bool verbose)
{
	float time;
	hipEvent_t start, stop;

	glm::vec3* d_positions = gameState.GetDevicePositions();
	unsigned int nrEntities = gameState.GetNrEntities();

	if (verbose) {
		printf("Starting physics with %d entities\n", nrEntities);
	}

	CubDebugExit(hipEventCreate(&start));
	CubDebugExit(hipEventCreate(&stop));

	CubDebugExit(hipEventRecord(start, 0));

	gameState.UpdateDevice();

	CubDebugExit(hipEventRecord(stop, 0));
	CubDebugExit(hipEventSynchronize(stop));
	CubDebugExit(hipEventElapsedTime(&time, start, stop));

	if (verbose) {
		printf("Time to update device buffers:  %3.3f ms \n", time);
	}

	CubDebugExit(hipEventRecord(start, 0));

	m_bvh.Build(d_positions, nrEntities);

	CubDebugExit(hipEventRecord(stop, 0));
	CubDebugExit(hipEventSynchronize(stop));
	CubDebugExit(hipEventElapsedTime(&time, start, stop));

	if (verbose) {
		printf("Time to build BVH:  %3.3f ms \n", time);
	}

	CubDebugExit(hipEventRecord(start, 0));

	m_bvh.BroadPhase(d_positions, nrEntities);

	CubDebugExit(hipEventRecord(stop, 0));
	CubDebugExit(hipEventSynchronize(stop));
	CubDebugExit(hipEventElapsedTime(&time, start, stop));

	if (verbose) {
		printf("Time for physics broad phase:  %3.3f ms \n", time);
	}

	CubDebugExit(hipEventRecord(start, 0));

	m_bvh.NarrowPhase(gameState.GetDeviceOldPositions(), d_positions, gameState.GetDeviceImpulses(), gameState.GetDeviceCorrections(), gameState.GetDeviceCollisionsNr(), nrEntities);

	CubDebugExit(hipEventRecord(stop, 0));
	CubDebugExit(hipEventSynchronize(stop));
	CubDebugExit(hipEventElapsedTime(&time, start, stop));

	if (verbose) {
		printf("Time for physics narrow phase:  %3.3f ms \n", time);
	}

	CubDebugExit(hipEventRecord(start, 0));

	gameState.ApplyForces();

	CubDebugExit(hipEventRecord(stop, 0));
	CubDebugExit(hipEventSynchronize(stop));
	CubDebugExit(hipEventElapsedTime(&time, start, stop));

	if (verbose) {
		printf("Time to apply forces:  %3.3f ms \n", time);
	}

	// m_bvh.NrCollisions(nrEntities);
}


